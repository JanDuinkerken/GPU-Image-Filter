#include "hip/hip_runtime.h"
/*
    Instructions
Develop a CUDA implementation of an image convolutional algorithm for RGB or RGBA images
(RGB or RGBA it depends on the image format). This assignment can be accomplished in group (up to 2 students).

You can refer to the file format you prefer, for instance PNG, JPG, or TIFF. I suggest to use a software library
for loading (into a buffer)/saving images like libpng (http://www.libpng.org/pub/png/libpng.html). libpng is already
installed on the JPDM2 workstation.
The kernels must receive the image as a linear buffer representing the pixels color in the RBG or RGBA format, together
with the convolutional filter (e.g., the one reported below), apply the filter, and store the resulting image into an
output buffer. Eventually, the filtered image must be saved to disk for assessment purposes.

Sharpen convolutional filter (ref: https://en.wikipedia.org/wiki/Kernel_(image_processing))

 0 -1  0
-1  5 -1
 0 -1  0


Assessment

Run some experiments by using three block sizes, namely 8x8, 8x16, 16x8, 16x16, 16x32, 32x16 and 32x32 by profiling
the executions into a table reporting the elapsed times and the bytes accessed L1, L2 and DRAM memory systems.
*/

// TODO: Test it with the different grid sizes (args)
// TODO: Test the performance with the nvprof

#include <stdio.h>
#include <stdint.h>
#include "../utils/spng/spng.h"

#include "../utils/utils.h"

// Input and output paths
#define INPATH "../input/pngtest.png"
#define OUTPATH "../output/sharpened_pngtest.png"

#define SHARPEN_SIZE 1
#define F_EXPANSION (SHARPEN_SIZE * 2 + 1)
#define F_PITCH 3
#define COLOR_VALUES 4

__global__ void sharpenFilterKernel(unsigned char *d_image, unsigned char *d_mod_image, int offset,
                                    int filter[F_EXPANSION][F_EXPANSION], int width, int height)
{
    int column = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (column < width && row < height)
    {
        for (int sharpenRow = -SHARPEN_SIZE; sharpenRow < SHARPEN_SIZE + 1; sharpenRow++)
            for (int sharpenCol = -SHARPEN_SIZE; sharpenCol < SHARPEN_SIZE + 1; sharpenCol++)
            {
                int currentRow = row + sharpenRow;
                int currentColumn = column + sharpenCol;
                if (currentRow > -1 && currentRow < height && currentColumn > -1 && currentColumn < width)
                {
                    d_mod_image[COLOR_VALUES * (row * width + column)    ]
                        += d_image[COLOR_VALUES * (currentRow * width + currentColumn)    ]
                            * filter[sharpenRow][sharpenCol];  // R

                    d_mod_image[COLOR_VALUES * (row * width + column) + 1]
                        += d_image[COLOR_VALUES * (currentRow * width + currentColumn) + 1]
                            * filter[sharpenRow][sharpenCol];  // G

                    d_mod_image[COLOR_VALUES * (row * width + column) + 2]
                        += d_image[COLOR_VALUES * (currentRow * width + cucurrentColumnrCol) + 2]
                            * filter[sharpenRow][sharpenCol];  // B

                    d_mod_image[COLOR_VALUES * (row * width + column) + 3]
                        += d_image[COLOR_VALUES * (currentRow * width + currentColumn) + 3]
                            * filter[sharpenRow][sharpenCol];  // A
                }
            }
    }
}

// for error-handling on operations that return hipError_t
void checkReturnedError(hipError_t error, int line)
{
    if (error != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(error), __FILE__, line);
        // hipDeviceReset();
        exit(EXIT_FAILURE);
    }
}

void process_image(size_t image_size, unsigned char *image, unsigned char *mod_image,
                   int block_size_x, int block_size_y, int n, int h_height, int h_width)
{
    unsigned char *d_mod_image;
    unsigned char *d_image;
    dim3 block_size(block_size_x, block_size_y, 1);
    dim3 grid_size((int)ceil((float)n / block_size_x), (int)ceil((float)n / block_size_y), 1);

    hipError_t error = hipMalloc(&d_image, image_size);
    checkReturnedError(error, __LINE__);
    error = hipMalloc(&d_mod_image, image_size);
    checkReturnedError(error, __LINE__);
    error = hipMemset(&d_mod_image, 0, image_size);
    checkReturnedError(error, __LINE__);

    error = hipMemcpy(d_image, image, image_size, hipMemcpyHostToDevice);
    checkReturnedError(error, __LINE__);

    // Sharpen convolutional filter
    int filter[F_EXPANSION][F_EXPANSION] = {{0, -1, 0}, {-1, 5, -1}, {0, -1, 0}};
    int d_filter[F_EXPANSION][F_EXPANSION];
    hipMemcpy2D(d_filter, F_PITCH, filter, F_PITCH, F_EXPANSION * sizeof(int), F_EXPANSION * sizeof(int), hipMemcpyHostToDevice);

    int *width;
    int *height;

    hipMallocManaged(&width, sizeof(int));
    hipMallocManaged(&height, sizeof(int));

    *width = h_width;
    *height = h_height;

    for (int i = 0; i < 4; i++)
        sharpenFilterKernel<<<grid_size, block_size>>>(d_image, d_mod_image, i, d_filter, *width, *height);

    error = hipMemcpy(mod_image, d_mod_image, image_size, hipMemcpyDeviceToHost);
    checkReturnedError(error, __LINE__);

    hipFree(d_mod_image);
    hipFree(d_image);
}

int main(int argc, char **argv)
{
    spng_ctx *ctx = NULL;
    struct spng_ihdr ihdr;
    spng_color_type color_type;
    size_t image_size, image_width;
    unsigned char *image = NULL;
    unsigned char *mod_image = NULL;
    int block_size_x, block_size_y, n;
    block_size_x = atoi(argv[1]);
    block_size_y = atoi(argv[2]);
    n = atoi(argv[3]);

    if (decode_png(INPATH, ctx, &ihdr, &image, &image_size, &image_width, &color_type))
        return 1;

    mod_image = (unsigned char *)malloc(image_size);
    if (mod_image == NULL)
    {
        printf("Error allocating the necessary memory to store the modified image");
        return 1;
    }
    process_image(image_size, image, mod_image, block_size_x, block_size_y, n, ihdr.height, image_width);

    encode_png(mod_image, image_size, ihdr.width, ihdr.height, color_type, ihdr.bit_depth, OUTPATH);

    spng_ctx_free(ctx);
    free(image);
    free(mod_image);

    return 0;
}
