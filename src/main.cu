#include "hip/hip_runtime.h"
/*
    Instructions
Develop a CUDA implementation of an image convolutional algorithm for RGB or
RGBA images (RGB or RGBA it depends on the image format). This assignment can be
accomplished in group (up to 2 students).

You can refer to the file format you prefer, for instance PNG, JPG, or TIFF. I
suggest to use a software library for loading (into a buffer)/saving images like
libpng (http://www.libpng.org/pub/png/libpng.html). libpng is already installed
on the JPDM2 workstation. The kernels must receive the image as a linear buffer
representing the pixels color in the RBG or RGBA format, together with the
convolutional filter (e.g., the one reported below), apply the filter, and store
the resulting image into an output buffer. Eventually, the filtered image must
be saved to disk for assessment purposes.

Sharpen convolutional filter (ref:
https://en.wikipedia.org/wiki/Kernel_(image_processing))

 0 -1  0
-1  5 -1
 0 -1  0


Assessment

Run some experiments by using three block sizes, namely 8x8, 8x16, 16x8, 16x16,
16x32, 32x16 and 32x32 by profiling the executions into a table reporting the
elapsed times and the bytes accessed L1, L2 and DRAM memory systems.
*/

#include "../utils/spng/spng.h"
#include <stdint.h>
#include <stdio.h>

#include "../utils/utils.h"

// Input and output paths
#define INPATH "../input/pngtest.png"
#define OUTPATH "../output/sharpened_pngtest.png"

#define SHARPEN_SIZE (1)
// assume a quadratic filter
#define F_EXPANSION (SHARPEN_SIZE * 2 + 1)
#define F_PITCH (3)
#define COLOR_VALUES (4)

__global__ void sharpenFilterKernel(unsigned char *d_image,
                                    unsigned char *d_mod_image, int *filter,
                                    int width, int height) {
  int column = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  if (column < width && row < height) {
    for (int sharpenRow = -SHARPEN_SIZE; sharpenRow < SHARPEN_SIZE + 1;
         sharpenRow++)
      for (int sharpenCol = -SHARPEN_SIZE; sharpenCol < SHARPEN_SIZE + 1;
           sharpenCol++) {
        int currentRow = row + sharpenRow;
        int currentColumn = column + sharpenCol;
        if (currentRow > -1 && currentRow < height && currentColumn > -1 &&
            currentColumn < width) {
          d_mod_image[COLOR_VALUES * (row * width + column)] // R
              += d_image[COLOR_VALUES * (currentRow * width + currentColumn)] *
                 filter[sharpenRow * width + sharpenCol];

          d_mod_image[COLOR_VALUES * (row * width + column) + 1] // G
              +=
              d_image[COLOR_VALUES * (currentRow * width + currentColumn) + 1] *
              filter[sharpenRow * width + sharpenCol];

          d_mod_image[COLOR_VALUES * (row * width + column) + 2] // B
              +=
              d_image[COLOR_VALUES * (currentRow * width + currentColumn) + 2] *
              filter[sharpenRow * width + sharpenCol];

          d_mod_image[COLOR_VALUES * (row * width + column) +
                      3] // A --> We do not apply the filter here
              = d_image[COLOR_VALUES * (currentRow * width + currentColumn) +
                        3];
        }
      }
  }
}

// for error-handling on operations that return hipError_t
void checkReturnedError(hipError_t error, int line) {
  if (error != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(error), __FILE__, line);
    // hipDeviceReset();
    exit(EXIT_FAILURE);
  }
}

// for error-handling on operations that do not return any error
void checkError(int line) {
  hipError_t error = hipGetLastError();
  checkReturnedError(error, line);
}

void process_image(size_t image_size, unsigned char *image,
                   unsigned char *mod_image, int block_size_x, int block_size_y,
                   int h_height, int h_width) {
  unsigned char *d_mod_image;
  unsigned char *d_image;
  dim3 block_size(block_size_x, block_size_y, 1);
  dim3 grid_size((int)ceil((float)h_width / block_size_x),
                 (int)ceil((float)h_height / block_size_y), 1);

  hipError_t error = hipMalloc(&d_image, image_size);
  checkReturnedError(error, __LINE__);
  error = hipMalloc(&d_mod_image, image_size);
  checkReturnedError(error, __LINE__);
  error = hipMemset(d_mod_image, 0, image_size);
  checkReturnedError(error, __LINE__);

  error = hipMemcpy(d_image, image, image_size, hipMemcpyHostToDevice);
  checkReturnedError(error, __LINE__);

  // Sharpen convolutional filter
  int filter[F_EXPANSION * F_EXPANSION] = {0, -1, 0, -1, 5, -1, 0, -1, 0};
  int *d_filter;
  error = hipMalloc(&d_filter, F_EXPANSION * F_EXPANSION * sizeof(int));
  checkReturnedError(error, __LINE__);
  error = hipMemcpy(d_filter, filter, F_EXPANSION * F_EXPANSION * sizeof(int),
                     hipMemcpyHostToDevice);
  checkReturnedError(error, __LINE__);

  int *width;
  int *height;
  error = hipMallocManaged(&width, sizeof(int));
  checkReturnedError(error, __LINE__);
  error = hipMallocManaged(&height, sizeof(int));
  checkReturnedError(error, __LINE__);
  *width = h_width;
  *height = h_height;

  sharpenFilterKernel<<<grid_size, block_size>>>(d_image, d_mod_image, d_filter,
                                                 *width, *height);
  checkError(__LINE__);

  // Illegal memory access when using pngtest_2.png
  error =
      hipMemcpy(mod_image, d_mod_image, image_size, hipMemcpyDeviceToHost);
  checkReturnedError(error, __LINE__);

  hipFree(d_mod_image);
  hipFree(d_image);
  hipFree(d_filter);
  hipFree(width);
  hipFree(height);
}

int main(int argc, char **argv) {
  spng_ctx *ctx = NULL;
  struct spng_ihdr ihdr;
  spng_color_type color_type;
  size_t image_size, image_width;
  unsigned char *image = NULL;
  unsigned char *mod_image = NULL;
  int block_size_x, block_size_y;
  block_size_x = atoi(argv[1]);
  block_size_y = atoi(argv[2]);
  unsigned char *expanded_image = NULL;

  if (decode_png(INPATH, ctx, &ihdr, &image, &image_size, &image_width,
                 &color_type))
    return 1;

  mod_image = (unsigned char *)malloc(image_size);
  if (mod_image == NULL) {
    printf("Error allocating the necessary memory to store the modified image");
    return 1;
  }

  size_t expanded_size =
      image_size +
      sizeof(unsigned char) * (2 * (image_width + 8) + ihdr.height * 8);
  expanded_image = (unsigned char *)malloc(expanded_size);
  expand_image(expanded_image, image, image_size, image_width, ihdr.height);

  process_image(image_size, image, mod_image, block_size_x, block_size_y,
                ihdr.height, image_width);

  encode_png(mod_image, image_size, ihdr.width, ihdr.height, color_type,
             ihdr.bit_depth, OUTPATH);

  spng_ctx_free(ctx);
  free(image);
  free(mod_image);

  return 0;
}
